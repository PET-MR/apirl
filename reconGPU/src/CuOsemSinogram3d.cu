#include "hip/hip_runtime.h"
#include <CuOsemSinogram3d.h>

// Memoria constante con los valores de los angulos de la proyeccion,
extern __device__ __constant__ float d_thetaValues_deg[MAX_PHI_VALUES];

// Memoria constante con los valores de la distancia r.
extern __device__ __constant__ float d_RValues_mm[MAX_R_VALUES];

// Memoria constante con los valores de la coordenada axial o z.
extern __device__ __constant__ float d_AxialValues_mm[MAX_Z_VALUES];

extern __device__ __constant__ float d_RadioScanner_mm;

extern __device__ __constant__ float d_AxialFov_mm;

extern __device__ __constant__ float d_RadioFov_mm;

extern __device__ __constant__ SizeImage d_imageSize;

extern __device__ __constant__ int d_numPixels;

extern __device__ __constant__ int d_numPixelsPerSlice;

extern __device__ __constant__ int d_numBinsSino2d;

extern texture<float, 3, hipReadModeElementType> texImage;  // 3D texture

extern surface<void, 3> surfImage;

CuOsemSinogram3d::CuOsemSinogram3d(Sinogram3D* cInputProjection, Image* cInitialEstimate, string cPathSalida, string cOutputPrefix, int cNumIterations, int cSaveIterationInterval, bool cSaveIntermediate, bool cSensitivityImageFromFile, CuProjector* cForwardprojector, CuProjector* cBackprojector, int cNumSubsets) : CuMlemSinogram3d(cInputProjection, cInitialEstimate, cPathSalida, cOutputPrefix, cNumIterations, cSaveIterationInterval, cSaveIntermediate, cSensitivityImageFromFile, cForwardprojector, cBackprojector)
{
  numSubsets = cNumSubsets;
  // Tengo que crear la sensitivity images y el vector de thresholds:
  updateThresholds = (float*) malloc(sizeof(float)*numSubsets);
}

CuOsemSinogram3d::CuOsemSinogram3d(string configFilename):CuMlemSinogram3d(configFilename)
{
    /// Inicializo las variables con sus valores por default
    
}


bool CuOsemSinogram3d::InitGpuMemory(TipoProyector tipoProy)
{
  // Número total de píxeles.
  int numPixels = reconstructionImage->getPixelCount();
  // Número total de bins del sinograma:
  int numBins = inputProjection->getBinCount();
  // Número total de bins por subset, que es el que voy a tener que usar para los proyectores:
  int numBinsSubset = inputProjection->getSubset(0, numSubsets)->getBinCount();
  // Lo mismo para el numero de sinogramas:
  int numSinograms = inputProjection->getNumSinograms();
  // Pido memoria para la gpu, debo almacenar los sinogramas y las imágenes.
  // Lo hago acá y no en el proyector para mantenerme en memmoria de gpu durante toda la reconstrucción.
  
  checkCudaErrors(hipMalloc((void**) &d_reconstructionImage, sizeof(float)*numPixels));
  checkCudaErrors(hipMalloc((void**) &d_backprojectedImage, sizeof(float)*numPixels));
  // Para la proyección estimada siempre va a ser del tamaño del subset.
  checkCudaErrors(hipMalloc((void**) &d_estimatedProjection, sizeof(float)*numBinsSubset));
  if(enableAdditiveTerm)
    checkCudaErrors(hipMalloc((void**) &d_additiveSinogram, sizeof(float)*numBinsSubset));
  checkCudaErrors(hipMalloc((void**) &d_ring1, sizeof(float)*inputProjection->getNumSinograms()));
  checkCudaErrors(hipMalloc((void**) &d_ring2, sizeof(float)*inputProjection->getNumSinograms()));
  // Para la proyección de entrada pido memoria, pero en caso de altar se podría sacar. Solo lo uso para 
  // obtener el likleihood.
  checkCudaErrors(hipMalloc((void**) &d_inputProjection, sizeof(float)*numBins));
  // Por ahora tengo las dos, d_ring1 me da el índice de anillo, y d_ring1_mm me da directamente la coordenada axial.
  // Agregue esto porque para usar una única LOR para
  checkCudaErrors(hipMalloc((void**) &d_ring1_mm, sizeof(int)*numSinograms));
  checkCudaErrors(hipMalloc((void**) &d_ring2_mm, sizeof(int)*numSinograms));
  // Para la sensitivity iamge, tengo un array de sensitivity images. El array de punteros tiene que estar en cpu, luego
  // cada uno de esos punteros tiene que tener una dirección de gpu.
  d_sensitivityImages = (float**)malloc(sizeof(float*)*numSubsets);	
  // Para los subsets lo mismo:
  d_inputProjectionSubsets = (float**)malloc(sizeof(float*)*numSubsets);
  // Ángulos de los subsets:
  h_subsetsAngles = (float**)malloc(sizeof(float*)*numSubsets);
  // Sinogram3d for store temporarily the subsets:
  Sinogram3D* subset;
  for(int i = 0; i < numSubsets; i++)
  {
    // Memoria para cada sensitivty image:
    checkCudaErrors(hipMalloc(d_sensitivityImages+i, sizeof(float)*numPixels));	// Memoria para las imagenes con los punteros de cpu y luego copio a gpu.
    // Pongo en cero la imágens de sensibilidad:
    checkCudaErrors(hipMemset(d_sensitivityImages[i], 0,sizeof(float)*numPixels));
    // Memoria para cada subset del sinograma de entrada:
    checkCudaErrors(hipMalloc(d_inputProjectionSubsets+i, sizeof(float)*numBinsSubset));
    // Copio el subset del sinograma de entrada, llamo a una función porque tengo que ir recorriendo todos los sinogramas:
    subset = inputProjection->getSubset(i, numSubsets);
    CopySinogram3dHostToGpu(d_inputProjectionSubsets[i], subset);
    // Inicio los ángulos de cada proyección:
    h_subsetsAngles[i] = (float*) malloc(sizeof(float)*subset->getNumProj());
    memcpy(h_subsetsAngles[i], subset->getAngPtr(), sizeof(float)*subset->getNumProj());
    numProj = subset->getNumProj();
    delete subset;
  }
  
  // Copio la iamgen inicial:
  checkCudaErrors(hipMemcpy(d_reconstructionImage, initialEstimate->getPixelsPtr(),sizeof(float)*numPixels,hipMemcpyHostToDevice));
  // Pongo en cero la imágen de retroproyección:
  checkCudaErrors(hipMemset(d_backprojectedImage, 0,sizeof(float)*numPixels));
  // Copio el sinograma de entrada, llamo a una función porque tengo que ir recorriendo todos los sinogramas:
  CopySinogram3dHostToGpu(d_inputProjection, inputProjection); // No se si realmente lo necesito.
  // Pongo en cero el sinograma de proyección:
  checkCudaErrors(hipMemset(d_estimatedProjection, 0,sizeof(float)*numBinsSubset));
  // Memoria para el valor de likelihood.
  checkCudaErrors(hipMalloc((void**) &d_likelihood, sizeof(float)));
  checkCudaErrors(hipMemset(d_likelihood, 0,sizeof(float)));
  // Además de copiar los valores de todos los bins, debo inicializar todas las constantes de reconstrucción.
  // Por un lado tengo los valores de coordenadas posibles de r, theta y z. Los mismos se copian a memoria constante de GPU (ver vectores globales al inicio de este archivo.
  // Los theta values los tengo que cargar por cada subset.
  // checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_thetaValues_deg), inputProjection->getAngPtr(), sizeof(float)*inputProjection->getNumProj()));
  int auxInt;
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_RValues_mm), inputProjection->getSegment(0)->getSinogram2D(0)->getRPtr(), sizeof(float)*inputProjection->getNumR()));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_AxialValues_mm), inputProjection->getAxialPtr(), sizeof(float)*inputProjection->getNumRings()));
  SizeImage size =  reconstructionImage->getSize();
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_imageSize), &size, sizeof(reconstructionImage->getSize())));
  float aux;
  aux = reconstructionImage->getFovRadio();
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_RadioFov_mm), &aux, sizeof(inputProjection->getRadioFov_mm())));
  aux = reconstructionImage->getFovHeight();
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_AxialFov_mm), &aux, sizeof(inputProjection->getAxialFoV_mm())));

  auxInt = size.nPixelsX * size.nPixelsY;
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_numPixelsPerSlice), &auxInt, sizeof(int)));
  auxInt = auxInt * size.nPixelsZ;
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_numPixels), &auxInt, sizeof(int)));
  auxInt = numBinsSubset / numSinograms;
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_numBinsSino2d), &auxInt, sizeof(int)));
  
  // Para el sinograma 3d tengo que cada sino 2d puede representar varios sinogramas asociados a distintas combinaciones de anillos.
  // En la versión con CPU proceso todas las LORs, ahora solo voy a considerar la del medio, que sería la ventaja de reducir el volumen de LORs.
  // Entonces genero un array con las coordenadas de anillos de cada sinograma.
  int iSino = 0;
  float* auxRings1 = (float*)malloc(sizeof(float)*numSinograms);
  float* auxRings2 = (float*)malloc(sizeof(float)*numSinograms);
  float* auxRings1_mm = (float*)malloc(sizeof(float)*numSinograms);
  float* auxRings2_mm = (float*)malloc(sizeof(float)*numSinograms);
  float numZ;
  for(int i = 0; i < inputProjection->getNumSegments(); i++)
  {
    for(int j = 0; j < inputProjection->getSegment(i)->getNumSinograms(); j++)
    {
      numZ = inputProjection->getSegment(i)->getSinogram2D(j)->getNumZ();
      // The ring is in fact the slice, goes from 1 to 2*numRings-1 (in c 0 to 2*numRings-2). For the real ring it would be (max+min)/2 bu since we want the slice we need to multiply by 2.
      auxRings1[iSino] = (inputProjection->getSegment(i)->getSinogram2D(j)->getRing1FromList(0)+inputProjection->getSegment(i)->getSinogram2D(j)->getRing1FromList(numZ-1));
      auxRings2[iSino] = (inputProjection->getSegment(i)->getSinogram2D(j)->getRing2FromList(0)+inputProjection->getSegment(i)->getSinogram2D(j)->getRing2FromList(numZ-1));
      // Es el promedio : cuando es par el index medio me da el índice menor pero con base 1, por eso le debo restar 1 para tener indices que inician en cero.
      auxRings1_mm[iSino] = (inputProjection->getSegment(i)->getSinogram2D(j)->getAxialValue1FromList(0) + inputProjection->getSegment(i)->getSinogram2D(j)->getAxialValue1FromList(numZ-1))/2;
      auxRings2_mm[iSino] = (inputProjection->getSegment(i)->getSinogram2D(j)->getAxialValue2FromList(0) + inputProjection->getSegment(i)->getSinogram2D(j)->getAxialValue2FromList(numZ-1))/2;
      iSino++;
    }
  }
  // Copio los índices de anillos a memoris de GPU:
  checkCudaErrors(hipMemcpy(d_ring1, auxRings1, sizeof(float)*inputProjection->getNumSinograms(), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_ring2, auxRings2, sizeof(float)*inputProjection->getNumSinograms(), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_ring1_mm, auxRings1_mm, sizeof(float)*numSinograms, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_ring2_mm, auxRings2_mm, sizeof(float)*numSinograms, hipMemcpyHostToDevice));

  // Datos que dependen del proyctor:
  switch(tipoProy)
  {
    case SIDDON_CYLINDRICAL_SCANNER:
      aux = ((Sinogram3DCylindricalPet*)inputProjection)->getRadioScanner_mm();
      checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_RadioScanner_mm), &aux, sizeof(float)));
      //checkCudaErrors(hipMemcpy(&d_RadioScanner_mm, &aux, sizeof(aux), hipMemcpyHostToDevice));
      break;
    case SIDDON_BACKPROJ_SURF_CYLINDRICAL_SCANNER:
      aux = ((Sinogram3DCylindricalPet*)inputProjection)->getRadioScanner_mm();
      checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_RadioScanner_mm), &aux, sizeof(float)));
      //checkCudaErrors(hipMemcpy(&d_RadioScanner_mm, &aux, sizeof(aux), hipMemcpyHostToDevice));
      break;
    case SIDDON_PROJ_TEXT_CYLINDRICAL_SCANNER:
      aux = ((Sinogram3DCylindricalPet*)inputProjection)->getRadioScanner_mm();
      checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_RadioScanner_mm), &aux, sizeof(float)));
      //checkCudaErrors(hipMemcpy(&d_RadioScanner_mm, &aux, sizeof(aux), hipMemcpyHostToDevice));
      break;
  }
  
  // Initialize texture (might be used with the some projectors):
  hipChannelFormatDesc floatTex = hipCreateChannelDesc<float>();
  const hipExtent extentImageSize = make_hipExtent(reconstructionImage->getSize().nPixelsX, reconstructionImage->getSize().nPixelsY, reconstructionImage->getSize().nPixelsZ);
  hipMemcpy3DParms copyParams = {0};
  // set texture parameters
  texImage.normalized = false;                      // access with normalized texture coordinates
  texImage.filterMode = hipFilterModeLinear;      // linear interpolation
  texImage.addressMode[0] = hipAddressModeBorder;   // wrap texture coordinates
  texImage.addressMode[1] = hipAddressModeBorder;
  texImage.addressMode[2] = hipAddressModeBorder;
  // The image is in a texture memory:  hipChannelFormatDesc floatTex;
  checkCudaErrors(hipMalloc3DArray(&d_imageArray, &floatTex, extentImageSize));
  // bind array to 3D texture
  checkCudaErrors(hipBindTextureToArray(texImage, d_imageArray, floatTex));
  
  // Libero memoria de vectores auxiliares:
  free(auxRings1);
  free(auxRings2);
  free(auxRings1_mm);
  free(auxRings2_mm);
  return true;
}

/// Actualiza los tamaños para los subsets.
void CuOsemSinogram3d::updateGridSizeForSubsetSinogram()
{
  unsigned int numBlocksX = 1, numBlocksY = 1, numBlocksZ = 1;
  // Con la dimensión x de la grilla completo el sino 2d:
  numBlocksX = ceil(( (float)(inputProjection->getNumProj() * inputProjection->getNumR()/numSubsets)  / blockSizeProjector.x));
  // La dimensión y, procesa cada sinograma. 
  numBlocksY = inputProjection->getNumSinograms();
  gridSizeProjector = dim3(numBlocksX, numBlocksY, numBlocksZ);
  // Con esta configuración seteo el proyector:
  forwardprojector->setKernelConfig(blockSizeProjector.x, blockSizeProjector.y, blockSizeProjector.z, numBlocksX, numBlocksY, numBlocksZ);
  
  // Con la dimensión x de la grilla completo el sino 2d:
  numBlocksX = ceil(((float)(inputProjection->getNumProj() * inputProjection->getNumR()/numSubsets) / blockSizeBackprojector.x));
  // La dimensión y, procesa cada sinograma. 
  numBlocksY = inputProjection->getNumSinograms();
  gridSizeBackprojector = dim3(numBlocksX, numBlocksY, numBlocksZ);
  backprojector->setKernelConfig(blockSizeBackprojector.x, blockSizeBackprojector.y, blockSizeBackprojector.z, numBlocksX, numBlocksY, numBlocksZ);
  
  // Update the number of bins:
  int numBinsSino2Subset = inputProjection->getNumProj() * inputProjection->getNumR()/numSubsets;
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_numBinsSino2d), &numBinsSino2Subset, sizeof(int)));
  
  #ifdef __DEBUG__
  printf("Block size for Subset Sinogram: %dx%dx%d. Grid Size: %dx%dx%d. Bins per subset: %d.\n",  blockSizeBackprojector.x,  blockSizeBackprojector.y,  blockSizeBackprojector.z, numBlocksX, numBlocksY, numBlocksZ, numBinsSino2Subset);
  #endif
}

/// Actualiza los tamaños para los subsets.
void CuOsemSinogram3d::updateGridSizeForWholeSinogram()
{
  unsigned int numBlocksX = 1, numBlocksY = 1, numBlocksZ = 1;
  // Con la dimensión x de la grilla completo el sino 2d:
  numBlocksX = ceil(((float)inputProjection->getNumProj() * inputProjection->getNumR()) / blockSizeProjector.x);
  // La dimensión y, procesa cada sinograma. 
  numBlocksY = inputProjection->getNumSinograms();
  gridSizeProjector = dim3(numBlocksX, numBlocksY, numBlocksZ);
  // Con esta configuración seteo el proyector:
  forwardprojector->setKernelConfig(blockSizeProjector.x, blockSizeProjector.y, blockSizeProjector.z, numBlocksX, numBlocksY, numBlocksZ);
  
  numBlocksX = ceil(((float)inputProjection->getNumProj() * inputProjection->getNumR()) / blockSizeBackprojector.x);
  // La dimensión y, procesa cada sinograma. 
  numBlocksY = inputProjection->getNumSinograms();
  gridSizeBackprojector = dim3(numBlocksX, numBlocksY, numBlocksZ);
  backprojector->setKernelConfig(blockSizeBackprojector.x, blockSizeBackprojector.y, blockSizeBackprojector.z, numBlocksX, numBlocksY, numBlocksZ);
  
  // Update the number of bins:
  int numBins = inputProjection->getNumProj()*inputProjection->getNumR();
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_numBinsSino2d), &numBins, sizeof(int)));
}

bool CuOsemSinogram3d::InitSubsetConstants(int indexSubset)
{
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_thetaValues_deg), h_subsetsAngles[indexSubset], sizeof(float)*numProj));
  #ifdef __DEBUG__
  printf("Angles for subset: %d.\n", indexSubset);
  for(int i=0; i<numProj; i++)
    printf("%f\t",  h_subsetsAngles[indexSubset][i]);
  printf("\n");
  #endif
  // Update grid size for subset:
  updateGridSizeForSubsetSinogram();
  return true;
}

// Método de reconstrucción que no se le indica el índice de GPU, incializa la GPU 0 por defecto.
bool CuOsemSinogram3d::Reconstruct(TipoProyector tipoProy)
{
  Reconstruct(tipoProy,0);
  return true;
}

/// Método público que realiza la reconstrucción en base a los parámetros pasados al objeto Mlem instanciado
bool CuOsemSinogram3d::Reconstruct(TipoProyector tipoProy, int indexGpu)
{
  string outputFilename;	// String para los nombres de los archivos de salida.
  int nPixels = reconstructionImage->getPixelCount();
  /// Proyección auxiliar, donde guardo el sinograma proyectado:
  Sinogram3D* estimatedProjection; // = new Sinogram3D(inputProjection);
  Sinogram3D *inputSubset;
  estimatedProjection=inputProjection->getSubset(0,numSubsets);
  int nBins = estimatedProjection->getBinCount();	// The number of bins that we use in the projection and backrpojection is the one of the subset.
  int nBinsSino2d = estimatedProjection->getNumProj() * estimatedProjection->getNumR();
  /// Inicializo el volumen a reconstruir con la imagen del initial estimate:
  reconstructionImage = new Image(initialEstimate);
  float* ptrPixels = reconstructionImage->getPixelsPtr();
  /// Change the grid size of the kernel to the size of the subsets:
  updateGridSizeForSubsetSinogram();
  /// Hago el log de la reconstrucción:
  Logger* logger = new Logger(logFileName);
  // INICALIZACIÓN DE GPU 
  if(!initCuda (0, logger))
  {
	  return false;
  }
  // Inicializo memoria de GPU:
  this->InitGpuMemory(tipoProy);
  /// Tamaño de la imagen:
  SizeImage sizeImage = reconstructionImage->getSize();
  
  /// Imagen donde guardo la backprojection.
  Image* backprojectedImage = new Image(reconstructionImage->getSize());
  /// Puntero a la sensitivity image.
  float* ptrSensitivityPixels;
  /// Puntero a la sensitivity image.
  float* ptrBackprojectedPixels = backprojectedImage->getPixelsPtr();
  /// Puntero del array con los tiempos de reconstrucción por iteración.
  float* timesIteration_mseg;
  float timesTotalIteration_seg;
  /// Puntero del array con los tiempos de backprojection por iteración.
  float* timesBackprojection_mseg;
  /// Puntero del array con los tiempos de forwardprojection por iteración.
  float* timesForwardprojection_mseg;
  /// Puntero del array con los tiempos de pixel update por iteración.
  float* timesPixelUpdate_mseg;
  /// String de c para utlizar en los mensajes de logueo.
  char c_string[512];
  /// Pido memoria para los arrays, que deben tener tantos elementos como iteraciones:
  timesIteration_mseg = (float*)malloc(sizeof(float)*this->numIterations*this->numSubsets);
  timesBackprojection_mseg = (float*)malloc(sizeof(float)*this->numIterations*this->numSubsets);
  timesForwardprojection_mseg = (float*)malloc(sizeof(float)*this->numIterations*this->numSubsets);
  timesPixelUpdate_mseg = (float*)malloc(sizeof(float)*this->numIterations*this->numSubsets);
  /// El vector de likelihood puede haber estado alocado previamente por lo que eso realloc. Tiene
  /// un elemento más porque el likelihood es previo a la actualización de la imagen, o sea que inicia
  /// con el initialEstimate y termina con la imagen reconstruida.
  if(this->likelihoodValues == NULL)
  {
    /// No había sido alocado previamente así que utilizo malloc.
    this->likelihoodValues = (float*)malloc(sizeof(float)*(this->numIterations +1));
  }
  else
  {
    /// Ya había sido alocado previamente, lo realoco.
    this->likelihoodValues = (float*)realloc(this->likelihoodValues, sizeof(float)*(this->numIterations + 1));
  }
  if(sensitivityImageFromFile)
  {
    /// Leo todas las imágenes:
    for(int s = 0; s < numSubsets; s++)
    {
      /// Leo las distintas imágnes de sensibilidad. Para osem el sensitivity filename, tiene que tener el
      /// prefijo de los nombres, y luego de les agrega un _%d, siendo %d el índice de subset.
      sprintf(c_string, "%s_subset_%d.h33", sensitivityFilename.c_str(), s);
      /// Leo el sensitivity volume desde el archivo
      sensitivityImage->readFromInterfile(c_string);
      sensitivityImage->forcePositive();
      // Copy to gpu:
      checkCudaErrors(hipMemcpy((void*)d_sensitivityImages[s]	, sensitivityImage->getPixelsPtr(),sizeof(float)*nPixels,hipMemcpyHostToDevice));
    }
  }
  else
  {
    /// Calculo todas los sensitivty volume, tengo tantos como subset. Sino alcancanzar la ram
    /// para almacenar todos, debería calcularlos dentro del for por cada iteración:
    for(int s = 0; s < numSubsets; s++)
    {
      /// Calculo el sensitivity volume
      if(computeSensitivity(sensitivityImage, s, tipoProy)==false)
      {
	strError = "Error al calcular la sensitivity Image.";
	return false;
      }
      // La guardo en disco.
      string sensitivityFileName = outputFilenamePrefix;
      sprintf(c_string, "_sensitivity_subset_%d", s);
      sensitivityFileName.append(c_string);
      sensitivityImage->writeInterfile((char*)sensitivityFileName.c_str());
      updateThresholds[s] = sensitivityImage->getMaxValue()*0.005;
    }
  }
  
  /// Escribo el título y luego los distintos parámetros de la reconstrucción:
  logger->writeLine("######## CUDA OS-EM Reconstruction #########");
  logger->writeValue("Name", this->outputFilenamePrefix);
  logger->writeValue("Type", "ML-EM");
  sprintf(c_string, "%d", this->numIterations);
  logger->writeValue("Iterations", c_string);
  logger->writeValue("Input Projections", "3D Sinogram");
  sprintf(c_string, "%d[r] x %d[ang]", inputProjection->getNumR(), inputProjection->getNumProj());
  logger->writeValue("Size of Sinogram2D",c_string);
  sprintf(c_string, "%d", inputProjection->getNumRings());
  logger->writeValue("Rings", c_string);
  sprintf(c_string, "%d", inputProjection->getNumSegments());
  logger->writeValue("Segments", c_string);
  sprintf(c_string, "%d", sizeReconImage.nDimensions);
  logger->writeValue("Image Dimensions", c_string);
  sprintf(c_string, "%d[x] x %d[y] x %d[z]", this->sizeReconImage.nPixelsX, this->sizeReconImage.nPixelsY, this->sizeReconImage.nPixelsZ);
  logger->writeValue("Image Size", c_string);
  sprintf(c_string, "%dx%dx%d %dx%dx%d", blockSizeProjector.x, blockSizeProjector.y, blockSizeProjector.z, gridSizeProjector.x, gridSizeProjector.y, gridSizeProjector.z);
  logger->writeValue("Projection Kernel Size", c_string);
  sprintf(c_string, "%dx%dx%d %dx%dx%d", blockSizeBackprojector.x, blockSizeBackprojector.y, blockSizeBackprojector.z, gridSizeBackprojector.x, gridSizeBackprojector.y, gridSizeBackprojector.z);
  logger->writeValue("Backprojection Kernel Size", c_string);
  sprintf(c_string, "%dx%dx%d %dx%dx%d", blockSizeImageUpdate.x, blockSizeImageUpdate.y, blockSizeImageUpdate.z, gridSizeImageUpdate.x, gridSizeImageUpdate.y, gridSizeImageUpdate.z);
  logger->writeValue("Pixel Update Kernel Size", c_string);
  // También se realiza un registro de los tiempos de ejecución:
  clock_t initialClock = clock();
  //Start with the iteration
  printf("Iniciando Reconstrucción...\n");
  /// Arranco con el log de los resultados:
  strcpy(c_string, "_______RECONSTRUCCION_______");
  logger->writeLine(c_string, strlen(c_string));
  /// Voy generando mensajes con los archivos creados en el log de salida.
  
  for(unsigned int t = 0; t < this->numIterations; t++)
  {
    printf("Iteration Nº: %d\n", t+1);
    timesTotalIteration_seg = 0;
    // Por cada iteración debo repetir la operación para todos los subsets.
    for(unsigned int s = 0; s < this->numSubsets; s++)
    {
      printf("\tsubiteration Nº: %d", s+1);
      clock_t initialClockIteration = clock();
      // Init constants for the subset.
      InitSubsetConstants(s);
      // Now I need to use the sinogram d_inputProjectionSubsets[s].
      /// Pongo en cero la proyección estimada, y hago la backprojection.
      checkCudaErrors(hipMemset(d_estimatedProjection, 0,sizeof(float)*nBins));
      /// Proyección de la imagen:
      switch(tipoProy)
      {
	case SIDDON_PROJ_TEXT_CYLINDRICAL_SCANNER: // This siddon implementation has only projection, so it uses the standard backprojection.
	  CopyDevImageToTexture(d_reconstructionImage, reconstructionImage->getSize()); // Copy the reconstruction iamge to texture.
	  forwardprojector->Project(d_reconstructionImage, d_estimatedProjection, d_ring1_mm, d_ring2_mm, reconstructionImage, (Sinogram3DCylindricalPet*)estimatedProjection, false); // Debo pasa el subset solo para tener el tamaño correcto de sino por eso uso estimated.
	  break;
	case SIDDON_CYLINDRICAL_SCANNER:
	  forwardprojector->Project(d_reconstructionImage, d_estimatedProjection, d_ring1_mm, d_ring2_mm, reconstructionImage, (Sinogram3DCylindricalPet*)estimatedProjection, false);
	  break;
      }
      clock_t finalClockProjection = clock();
      
      // The additive term in the forward model (the multiplicative is only take into account in the sensitivity image,
      // so the additive term need to be dividived by the multipicative factors previously):
      if(enableAdditiveTerm)
      {
	CopySinogram3dHostToGpu(d_additiveSinogram, additiveProjection->getSubset(s, numSubsets)); //copy the additive subset.
	addSinograms(d_estimatedProjection, d_additiveSinogram, nBinsSino2d, nBins);
      }
      
      /// Si quiero guardar la proyección intermedia, lo hago acá, porque luego en la backprojection se modifica para hacer el cociente entre entrada y estimada:
      if(saveIntermediateProjectionAndBackprojectedImage)
      {
	CopySinogram3dGpuToHost(estimatedProjection, d_estimatedProjection);
	sprintf(c_string, "%s_projection_iter_%d_subset_%d", outputFilenamePrefix.c_str(), t, s); /// La extensión se le agrega en write interfile.
	outputFilename.assign(c_string);
	estimatedProjection->writeInterfile((char*)outputFilename.c_str());
      }
      
      /// Pongo en cero la imagen de corrección, y hago la backprojection.
      checkCudaErrors(hipMemset(d_backprojectedImage, 0,sizeof(float)*nPixels));
      switch(tipoProy)
      {
	case SIDDON_PROJ_TEXT_CYLINDRICAL_SCANNER: // This siddon implementation has only projection, so it uses the standard backprojection.
	case SIDDON_CYLINDRICAL_SCANNER:
	  backprojector->DivideAndBackproject(d_inputProjectionSubsets[s], d_estimatedProjection, d_backprojectedImage, d_ring1_mm, d_ring2_mm, (Sinogram3DCylindricalPet*)estimatedProjection, backprojectedImage, false);
	  break;
      }
      if(saveIntermediateProjectionAndBackprojectedImage)
      {
	// Copio la imagen en gpu a cpu:
	checkCudaErrors(hipMemcpy(ptrBackprojectedPixels, d_backprojectedImage, sizeof(float)*reconstructionImage->getPixelCount(),hipMemcpyDeviceToHost)); 
	sprintf(c_string, "%s_backprojected_iter_%d_subset_%d", outputFilenamePrefix.c_str(), t, s); /// La extensión se le agrega en write interfile.
	outputFilename.assign(c_string);
	backprojectedImage->writeInterfile((char*)outputFilename.c_str());
      }
      clock_t finalClockBackprojection = clock();
      /// Actualización del Pixel
      this->updatePixelValue(s);
      /// Verifico
      if(saveIntermediateProjectionAndBackprojectedImage)
      {
	// Primero tengo que obtener la memoria de GPU:
	CopyReconstructedImageGpuToHost();
	sprintf(c_string, "%s_iter_%d_subset_%d", outputFilenamePrefix.c_str(), t,s); /// La extensión se le agrega en write interfile.
	outputFilename.assign(c_string);
	reconstructionImage->writeInterfile((char*)outputFilename.c_str());
	/// Termino con el log de los resultados:
	sprintf(c_string, "Imagen de iteración %d de subset %d guardada en: %s", t, s, outputFilename.c_str());
	logger->writeLine(c_string);
      }
      clock_t finalClockIteration = clock();
      printf("\ttot:%fsec\n", (float)(finalClockIteration-initialClockIteration)*1000/(float)CLOCKS_PER_SEC);
      /// Cargo los tiempos:
      timesIteration_mseg[this->numSubsets*t+s] = (float)(finalClockIteration-initialClockIteration)*1000/(float)CLOCKS_PER_SEC;
      timesBackprojection_mseg[this->numSubsets*t+s] = (float)(finalClockBackprojection-finalClockProjection)*1000/(float)CLOCKS_PER_SEC;
      timesForwardprojection_mseg[this->numSubsets*t+s] = (float)(finalClockProjection-initialClockIteration)*1000/(float)CLOCKS_PER_SEC;
      timesPixelUpdate_mseg[this->numSubsets*t+s] = (float)(finalClockIteration-finalClockBackprojection)*1000/(float)CLOCKS_PER_SEC;
      timesTotalIteration_seg += (float)(finalClockIteration-initialClockIteration)/(float)CLOCKS_PER_SEC;
      
      // Elimino el subset.
      /*delete inputSubset;
      delete estimatedProjection;*/
    }
    /// Verifico
    if(saveIterationInterval != 0)
    {
      if((t%saveIterationInterval)==0)
      {
	CopyReconstructedImageGpuToHost();
	sprintf(c_string, "%s_iter_%d", outputFilenamePrefix.c_str(), t); /// La extensión se le agrega en write interfile.
	string outputFilename;
	outputFilename.assign(c_string);
	reconstructionImage->writeInterfile((char*)outputFilename.c_str());
	/// Termino con el log de los resultados:
	sprintf(c_string, "Imagen de iteración %d guardada en: %s", t, outputFilename.c_str());
	logger->writeLine(c_string);
	
	/// Guardo el likelihood (Siempre va una iteración atrás, ya que el likelihhod se calcula a partir de la proyección
	/// estimada, que es el primer paso del algoritmo). Se lo calculo al sinograma
	/// proyectado, respecto del de entrada.
	this->likelihoodValues[t] = this->getLikelihoodValue(tipoProy);
      }
    }
    printf("End Iteration Nº: %d. Time: %f.\n", t+1, timesTotalIteration_seg);
    
  }

  clock_t finalClock = clock();
  sprintf(c_string, "%s_final", outputFilenamePrefix.c_str()); /// La extensión se le agrega en write interfile.
  outputFilename.assign(c_string);
  reconstructionImage->writeInterfile((char*)outputFilename.c_str());
  /// Termino con el log de los resultados:
  sprintf(c_string, "Imagen final guardada en: %s", outputFilename.c_str());
  logger->writeLine(c_string);
  /// Calculo la proyección de la última imagen para poder calcular el likelihood final:
  this->likelihoodValues[this->numIterations] = this->getLikelihoodValue(tipoProy);

  float tiempoTotal = (float)(finalClock - initialClock)*1000/(float)CLOCKS_PER_SEC;
  /// Termino con el log de los resultados:
  strcpy(c_string, "_______RESULTADOS DE RECONSTRUCCION_______");
  logger->writeLine(c_string, strlen(c_string));
  sprintf(c_string, "%f", tiempoTotal);
  logger->writeValue("Tiempo Total de Reconstrucción:", c_string);
  /// Ahora guardo los tiempos por iteración y por etapa, en fila de valores.
  strcpy(c_string, "Tiempos de Reconstrucción por Iteración [mseg]");
  logger->writeLine(c_string, strlen(c_string));
  logger->writeRowOfNumbers(timesIteration_mseg, this->numIterations);
  strcpy(c_string, "Tiempos de Forwardprojection por Iteración [mseg]");
  logger->writeLine(c_string, strlen(c_string));
  logger->writeRowOfNumbers(timesForwardprojection_mseg, this->numIterations*this->numSubsets);
  strcpy(c_string, "Tiempos de Backwardprojection por Iteración [mseg]");
  logger->writeLine(c_string, strlen(c_string));
  logger->writeRowOfNumbers(timesBackprojection_mseg, this->numIterations*this->numSubsets);
  strcpy(c_string, "Tiempos de UpdatePixel por Iteración [mseg]");
  logger->writeLine(c_string, strlen(c_string));
  logger->writeRowOfNumbers(timesPixelUpdate_mseg, this->numIterations*this->numSubsets);
  /// Por último registro los valores de likelihood:
  strcpy(c_string, "Likelihood por Iteración:");
  logger->writeLine(c_string, strlen(c_string));
  logger->writeRowOfNumbers(this->likelihoodValues, this->numIterations + 1);

  /// Libero la memoria de los arrays:
  free(timesIteration_mseg);
  free(timesBackprojection_mseg);
  free(timesForwardprojection_mseg);
  free(timesPixelUpdate_mseg);
  delete estimatedProjection;
  return true;
}

bool CuOsemSinogram3d::updatePixelValue(int subset)
{
  // Llamo al kernel que actualiza el pixel.
  cuUpdatePixelValue<<<gridSizeImageUpdate, blockSizeImageUpdate>>>(d_reconstructionImage, d_backprojectedImage, d_sensitivityImages[subset], reconstructionImage->getSize(), updateThresholds[subset]);
  hipDeviceSynchronize();
  return true;
}


float CuOsemSinogram3d::getLikelihoodValue(TipoProyector tipoProy)
{
  float likelihood;
  float* d_auxProjection;
  checkCudaErrors(hipMemset(d_likelihood, 0,sizeof(float)));
  checkCudaErrors(hipMalloc((void**) &d_auxProjection, sizeof(float)*inputProjection->getBinCount()));
  /// Change the kernel size to the whole sinogram:
  updateGridSizeForWholeSinogram();
  /// Proyección de la imagen:
  switch(tipoProy)
  {
    case SIDDON_PROJ_TEXT_CYLINDRICAL_SCANNER: // This siddon implementation has only projection, so it uses the standard backprojection.
      CopyDevImageToTexture(d_reconstructionImage, reconstructionImage->getSize()); // Copy the reconstruction iamge to texture.
      forwardprojector->Project(d_reconstructionImage, d_auxProjection, d_ring1_mm, d_ring2_mm, reconstructionImage, (Sinogram3DCylindricalPet*)inputProjection, false); // Debo pasa el subset para tener el tamaño correcto de sino.
      break;
    case SIDDON_CYLINDRICAL_SCANNER:
      forwardprojector->Project(d_reconstructionImage, d_auxProjection, d_ring1_mm, d_ring2_mm, reconstructionImage, (Sinogram3DCylindricalPet*)inputProjection, false);
      break;
  }
  if(saveIntermediateProjectionAndBackprojectedImage)
  {
    Sinogram3D* estimatedProjection = inputProjection->Copy();
    string outputFilename;
    char c_string[200];
    CopySinogram3dGpuToHost(estimatedProjection, d_auxProjection);
    sprintf(c_string, "%s_projection_likelihood", outputFilenamePrefix.c_str()); /// La extensión se le agrega en write interfile.
    outputFilename.assign(c_string);
    estimatedProjection->writeInterfile((char*)outputFilename.c_str());
    delete estimatedProjection;
  }
      
  cuGetLikelihoodValue<<<gridSizeProjector, blockSizeProjector>>>(d_auxProjection, d_inputProjection, d_likelihood, inputProjection->getNumR(), inputProjection->getNumProj(), inputProjection->getNumRings(), inputProjection->getNumSinograms());
  /// Sincronización de todos los threads.
  hipDeviceSynchronize();
  checkCudaErrors(hipMemcpy(&likelihood, d_likelihood,sizeof(float),hipMemcpyDeviceToHost));
  checkCudaErrors(hipFree(d_auxProjection));
  return likelihood;
}

bool CuOsemSinogram3d::computeSensitivity(Image* outputImage, int indexSubset, TipoProyector tipoProy)
{  
  /// Creo un Sinograma ·D igual que el de entrada.
  Sinogram3D* constantSinogram3D;
  /// With normalization use the norm sinogram if not a constant sinogram:
  if (enableMultiplicativeTerm)
    constantSinogram3D = multiplicativeProjection->getSubset(indexSubset, numSubsets);
  else
  {
    constantSinogram3D = inputProjection->getSubset(indexSubset, numSubsets);
    constantSinogram3D->FillConstant(1);
  }
  /// Copio a gpu:
  CopySinogram3dHostToGpu(d_estimatedProjection, constantSinogram3D);
  // Init subset constants:
  InitSubsetConstants(indexSubset);
  /// Change the grid size of the kernel to the size of the subsets:
  updateGridSizeForSubsetSinogram();
  /// Por último hago la backprojection
  switch(tipoProy)
  {
    case SIDDON_PROJ_TEXT_CYLINDRICAL_SCANNER: // This siddon implementation has only projection, so it uses the standard backprojection.
    case SIDDON_CYLINDRICAL_SCANNER:
      backprojector->Backproject(d_estimatedProjection, d_sensitivityImages[indexSubset], d_ring1_mm, d_ring2_mm, (Sinogram3DCylindricalPet*)constantSinogram3D, reconstructionImage, false);
      // Copio la memoria de gpu a cpu, así se puede actualizar el umbral:
      checkCudaErrors(hipMemcpy(outputImage->getPixelsPtr(), d_sensitivityImages[indexSubset],sizeof(float)*outputImage->getPixelCount(),hipMemcpyDeviceToHost));
      break;
    case SIDDON_BACKPROJ_SURF_CYLINDRICAL_SCANNER:
      CopyDevImageToTexture(d_sensitivityImages[indexSubset], outputImage->getSize());
      backprojector->Backproject(d_estimatedProjection, d_sensitivityImages[indexSubset], d_ring1_mm, d_ring2_mm, (Sinogram3DCylindricalPet*)constantSinogram3D, reconstructionImage, false);
      CopyTextureToDevtImage(d_sensitivityImages[indexSubset], outputImage->getSize());
      CopyTextureToHostImage(outputImage);
      break;
  }
  delete constantSinogram3D;
  return true;
}